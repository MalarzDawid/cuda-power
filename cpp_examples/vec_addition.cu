
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>

__global__ void vectorAdd(int *a, int *b, int *c, int N) {
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int N = 1 << 16;

    // Host vector pointers
    int *h_a, *h_b, *h_c;
    size_t bytes = sizeof(int) * N;

    hipHostMalloc(&h_a, bytes, hipHostMallocDefault);
    hipHostMalloc(&h_b, bytes, hipHostMallocDefault);
    hipHostMalloc(&h_c, bytes, hipHostMallocDefault);

    // Initizalize vectors A & B
    for (int i = 0; i < N; i++) {
        h_a[i] = rand() % 100;
        h_b[i] = rand() % 100;
    }

    // Alocate memory on the device
    int *d_a, *d_b, *d_c;

    // Alocate device memory
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Copy data (CPU -> GPU)
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    // 1<<10 => 1024
    int NUM_THREADS = 1 << 10;
    int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS; // for example: (65536 + 1024 - 1) / 1024 => 64.99 ??

    // Run kernel
    vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c, N);
    
    // Copy output to host
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // Verify results
    for (int i = 0; i < N; i++) {
        assert(h_c[i] == h_a[i] + h_b[i]);
    }

    // Free pinned memory
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    std::cout << "Everything is ok" << std::endl;
}